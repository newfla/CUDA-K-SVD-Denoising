#include <svdLib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace svd;

void CuSolverGeSvdJ::init(Matrix* matrix){

    //Call parent method
    SvdCudaEngine::init(matrix);

    //Configuration gesvdj
    hipsolverDnCreateGesvdjInfo(&gesvdjParams);
    //hipsolverDnXgesvdjSetTolerance(gesvdjParams, tolerance);
    //hipsolverDnXgesvdjSetMaxSweeps(gesvdjParams, maxSweeps);

    //Allocate Space on device
    hipsolverDnSgesvdj_bufferSize(
        cusolverH, 
        jobZ,
        econ,
        input->m,
        input->n,
        deviceA,
        input->ld,
        deviceS,
        deviceU,
        input->m,
        deviceVT,
        input->n,
        &lWork,
        gesvdjParams);
        
    hipMalloc((void**) &deviceWork , sizeof(double)*lWork);
}

void CuSolverGeSvdJ::work(){

    //DGESVDJ
    hipsolverDnSgesvdj(
        cusolverH,
        jobZ,
        econ,
        input->m,
        input->n,
        deviceA,
        input->ld,
        deviceS,
        deviceU,
        input->m,
        deviceVT,
        input->n,
        deviceWork,
        lWork,
        deviceInfo,
        gesvdjParams
    );
    hipDeviceSynchronize();
    printStat();
}

std::vector<Matrix*> CuSolverGeSvdJ::getOutputMatrices(){
    hipFree(deviceInfo);
    hipsolverDnDestroyGesvdjInfo(gesvdjParams);
    return SvdCudaEngine::getOutputMatrices(); 
}

void CuSolverGeSvdJ::printStat(){
    double residual = 0;
    int executedSweeps = 0;

    hipsolverDnXgesvdjGetSweeps(
        cusolverH,
        gesvdjParams,
        &executedSweeps);
    
    hipsolverDnXgesvdjGetResidual(
        cusolverH,
        gesvdjParams,
        &residual);

    std::cout<<"Residual |A - U*S*V**H|_F = "<<residual;
    std::cout<<"\nNumber of executed sweeps = "<<executedSweeps<<std::endl;
}