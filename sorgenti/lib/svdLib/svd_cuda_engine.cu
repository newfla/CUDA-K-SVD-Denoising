#include <svdLib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace svd;
using namespace baseUtl;
using namespace thrust;

SvdCudaEngine::SvdCudaEngine(){}

//*******************************************************************************************************
//  Save the vector on which SVD will be executed, create cuSolver additional data and move data on HOST
//  input:  + matrix (Matrix*) float, collum-major
//******************************************************************************************************
void SvdCudaEngine::init(Matrix* matrix){

    //Call parent method
    SvdEngine::init(matrix);

    //Create cusolverDn handle
    hipsolverDnCreate(&cusolverH) ;

    //Alocate space for cusolverDnInfo
    hipMalloc ((void**)&deviceInfo, sizeof(int));

    //Allocate memory on device
    less = matrix->m;
    if(less > matrix->n)
        less= matrix->n;

    hipMalloc((void**) &deviceU, (matrix->ld)*(matrix->m)*sizeof(float));
    hipMalloc((void**) &deviceS, (less)*sizeof(float));
    hipMalloc((void**) &deviceVT, (matrix->n)*(matrix->n)*sizeof(float));

    //Copy matrix on device
    if(matrix->deviceVector == NULL)
        matrix->deviceVector = new device_vector<float>(matrix->hostVector->begin(), matrix->hostVector->end());
    deviceA = raw_pointer_cast(matrix->deviceVector->data());
}


//******************************************************************
//  Obtain input matrix SVD decompisition and free DEVICE resources 
//  output:  + matrices (Matrix*) float, collum-major
//*****************************************************************
thrust::host_vector<Matrix*> SvdCudaEngine::getOutputMatrices(){

    float *hostU, *hostVT, *hostS;
    Matrix *outputU, *outputVT, *outputS;

    //Cpu matrix resource allocation
    hostU = new float[(input->m)*(input->m)]();
    hostVT = new float[(input->n)*(input->n)]();
    hostS = new float[input->n]();

    //Copy back to host
    hipMemcpy(hostU, deviceU, (input->ld)*(input->m)*sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy(hostVT, deviceVT, (input->n)*(input->n)*sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy(hostS, deviceS, (less)*sizeof(float), hipMemcpyDeviceToHost);

    //Allocate memory on host
    outputU = new Matrix(input->ld, input->m, input->m, hostU);
    outputVT = new Matrix(input->n, input->n, input->n, hostVT);
    outputS = new Matrix (1, less, 1, hostS);

    //Save SVD
    output.push_back(outputU);
    output.push_back(outputS);
    output.push_back(outputVT);

    //Cleaning cuda memory 
    hipFree(deviceA);
    hipFree(deviceU);
    hipFree(deviceVT);
    hipFree(deviceS);
    hipFree(deviceWork);
    hipsolverDnDestroy(cusolverH);
    input->deviceVector = NULL;

    hipDeviceReset();

    return output;
}

//******************************************************************
//  Obtain input matrix SVD decompisition and free DEVICE resources 
//  output:  + matrices (Matrix*) float, collum-major DEVICE
//*****************************************************************
thrust::device_vector<Matrix*> SvdCudaEngine::getDeviceOutputMatrices(){

    Matrix *outputU, *outputVT, *outputS;

    //Wrap raw pointer
    device_ptr<float> u(deviceU),
                    vt(deviceVT),
                    s(deviceS);

    //Allocate memory on host
    outputU = new Matrix(input->ld, input->m, input->m, new device_vector<float>(u, u + (input->ld * input->m)));
    outputVT = new Matrix(input->n, input->n, input->n, new device_vector<float>(vt, vt + (input->n * input->n)));
    outputS = new Matrix (1, input->n, 1, new device_vector<float>(s, s + input->n));

    //Save SVD
    output.push_back(outputU);
    output.push_back(outputS);
    output.push_back(outputVT);

    //Cleaning cuda memory 
    hipFree(deviceA);
    hipFree(deviceWork);
    hipsolverDnDestroy(cusolverH);
    input->deviceVector = NULL;

    hipDeviceReset();

    return output;
}