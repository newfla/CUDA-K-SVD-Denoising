#include <svdLib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace svd;

CuSolverGeSvd::CuSolverGeSvd(){}

//*******************************************************************************************************
//  Save the vector on which SVD will be executed, create cuSolver additional data and move data on HOST
//  input:  + matrix (Matrix*) float, collum-major
//******************************************************************************************************
void CuSolverGeSvd::init(Matrix* matrix){

    //Call parent method
    SvdCudaEngine::init(matrix);

    //Allocate WorkSpace on device
    hipsolverDnSgesvd_bufferSize(cusolverH, matrix->m, matrix->n, &lWork);
    hipMalloc((void**) &deviceWork , sizeof(double)*lWork);
    
}

//*****************************************
//  CuSOlver SVD decomposition (QR METHOD)
//****************************************
void CuSolverGeSvd::work(){

    //DGESVD
    hipsolverDnSgesvd(
        cusolverH,
        'A',
        'A',
        input->m,
        input->n,
        deviceA,
        input->ld,
        deviceS,
        deviceU,
        input->m,
        deviceVT,
        input->n,
        deviceWork,
        lWork,
        deviceRWork,
        deviceInfo
    );
    hipDeviceSynchronize();
}

//******************************************************************
//  Obtain input matrix SVD decompisition and free DEVICE resources 
//  output:  + matrices (Matrix*) float, collum-major
//*****************************************************************
thrust::host_vector<Matrix*> CuSolverGeSvd::getOutputMatrices(){
    
    hipFree(deviceInfo);
    if(deviceRWork != NULL )
        hipFree(deviceRWork);
    return SvdCudaEngine::getOutputMatrices(); 
}