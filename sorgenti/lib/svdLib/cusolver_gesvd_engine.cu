#include <svdLib.h>

using namespace svd;
using namespace baseUtl;

CuSolverGeSvd::CuSolverGeSvd(){}

//*********************************************************************************************************
//  Save the vector on which SVD will be executed, create cuSolver additional data and move data on DEVICE
//  input:  + matrix (Matrix*) float, collum-major
//********************************************************************************************************
void CuSolverGeSvd::init(Matrix* matrix){

    //Call parent method
    SvdCudaEngine::init(matrix);

    //Allocate WorkSpace on device
    hipsolverDnSgesvd_bufferSize(*cusolverH, matrix->m, matrix->n, &lWork);
    hipMalloc((void**) &deviceWork , sizeof(double)*lWork);
    
}

//*****************************************
//  CuSOlver SVD decomposition (QR METHOD)
//****************************************
void CuSolverGeSvd::work(){

    //DGESVD
    hipsolverDnSgesvd(
        *cusolverH,
        'A',
        'A',
        input->m,
        input->n,
        raw_pointer_cast(deviceA->data()),
        input->ld,
        raw_pointer_cast(deviceS->data()),
        raw_pointer_cast(deviceU->data()),
        input->m,
        raw_pointer_cast(deviceVT->data()),
        input->n,
        deviceWork,
        lWork,
        deviceRWork,
        deviceInfo
    );
    hipDeviceSynchronize();
}

//******************************************************************
//  Obtain input matrix SVD decompisition and free DEVICE resources 
//  output:  + matrices (Matrix*) float, collum-major HOST
//*****************************************************************
thrust::host_vector<Matrix*> CuSolverGeSvd::getOutputMatrices(){
    
    hipFree(deviceInfo);
    if(deviceRWork != NULL )
        hipFree(deviceRWork);
    return SvdCudaEngine::getOutputMatrices(); 
}

//******************************************************************
//  Obtain input matrix SVD decompisition and free DEVICE resources 
//  output:  + matrices (Matrix*) float, collum-major DEViCE
//*****************************************************************
thrust::host_vector<baseUtl::Matrix*> CuSolverGeSvd::getDeviceOutputMatrices(){

    hipFree(deviceInfo);
    if(deviceRWork != NULL )
        hipFree(deviceRWork);
    return SvdCudaEngine::getDeviceOutputMatrices(); 
}