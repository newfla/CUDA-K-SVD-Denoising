#include "hip/hip_runtime.h"
#include <denoisingLib.h>

using namespace denoising;
using namespace svd;
using namespace baseUtl;
using namespace matUtl;
using namespace thrust;
using namespace thrust::placeholders;


CudaKSvdDenoiser::CudaKSvdDenoiser(){
}

CudaKSvdDenoiser::~CudaKSvdDenoiser(){
    //TODO
}

//********************
//  Istantiate SvdObj
//*******************
void CudaKSvdDenoiser::buildSvdContainer(){
    switch (type)
        {
            case CUSOLVER_GESVD:
                svdContainer = new SvdContainer(SvdEngine::factory(CUSOLVER_GESVD));
                break;

            default:
            case CUSOLVER_GESVDJ:
                svdContainer = new SvdContainer(SvdEngine::factory(CUSOLVER_GESVDJ));
                break;
         
        }
}

//***********************************************************************************************************************
//  Load denoising save
//  output:  + status (signed char) 0 = done, -1 = image loading failed, -2 = denoising failed, -3 = image saving failed
//**********************************************************************************************************************
signed char CudaKSvdDenoiser::denoising(){

    if(!loadImage())
        return -1;    
    
    if(!internalDenoising())
        return -2;

    if(!saveImage())
        return -3;

    return 0;
}

//**************************
//  Load image
//  output:  + staus (bool)
//*************************
bool CudaKSvdDenoiser::loadImage(){

    //TODO
    return Denoiser::loadImage();
}

//**************************
//  Save image
//  output:  + staus (bool)
//*************************
bool CudaKSvdDenoiser::saveImage(){

    //TODO
    return Denoiser::saveImage();
}

//**************************
//  CUDA K-SVD implementation 
//  output:  + staus (bool)
//*************************
bool CudaKSvdDenoiser::internalDenoising(){

    auto start = std::chrono::steady_clock::now();

    //Divide image in square patches column major of fixed dims
    createPatches();

    //Init Dict
    initDictionary();

    //Start #iter K-SVD
    kSvd();

    //Rebuild originalImage
    createImage();

    auto end = std::chrono::steady_clock::now();
    timeElapsed->working = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count();

    return true;
}

//**********************************************************************************************
//  Divide image in square patches column major of fixed dims (patchSquareDime x patcSquareDim)
//*********************************************************************************************
void CudaKSvdDenoiser::createPatches(){

    std::cout<<"Create Patches"<<std::endl;

    auto start = std::chrono::steady_clock::now();

    int i, j;
    host_vector<float>* patches = new host_vector<float>();

    //Create patch division on host

    for(i = 0; i + patchSquareDim <= inputMatrix->n; i+= slidingPatch){ //n = ImageWidth

        for(j = 0; j + patchSquareDim <= inputMatrix->m; j+= slidingPatch){ // m = ImageHeight

            host_vector<float> patch;
            int startPatch = i * inputMatrix->m + j;

            for(int k = startPatch; k < startPatch + patchSquareDim * inputMatrix->m; k += inputMatrix->m)
                patch.insert(patch.end(), inputMatrix->hostVector->begin() + k, inputMatrix->hostVector->begin() + k + patchSquareDim);

            patches->insert(patches->end(), patch.begin(), patch.end()); 
        }  
    }

    i = patchSquareDim * patchSquareDim;
    j = patches->size() / i;
    noisePatches = new Matrix(i, j, i, patches);
    
    std::cout<<"    # Patches: "<<j<<"  Dim: "<<i<<std::endl;

    //Copy data on device
    noisePatches->copyOnDevice();

    auto end = std::chrono::steady_clock::now();
    std::cout<<"    # Time Elapsed : "<<std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count()/1000.<<" s"<<std::endl<<std::endl;
}

//*************************************************************************************************************
//  Init a dictionary using #atoms square patches column major of fixed dims (patchSquareDim x patcSquareDim)
//************************************************************************************************************
void CudaKSvdDenoiser::initDictionary(){

    std::cout<<"Init Dictionary"<<std::endl;

    auto start = std::chrono::steady_clock::now();
    int dim = patchSquareDim * patchSquareDim;
    device_vector<float> * dict = new device_vector<float>();

    //Copy Patches and normalization using norm2
    for (int i = 0; i < atoms; i++){
        
        //Copy a single patch
        dict->insert(dict->end(), noisePatches->deviceVector->begin() + (i * dim), noisePatches->deviceVector->begin() + ((i + 1) * dim));

        //Calculate norm
        float norm = sqrtf(transform_reduce(dict->begin() + (i * dim), dict->begin() + ((i+1) * dim), mySquare<float>(), 0, myPlus<float>()));

        //Normalize vector
        transform(dict->begin() + (i * dim), dict->begin() + ((i + 1) * dim), dict->begin() + (i * dim), _1/norm);
    }

    dictionary = new Matrix(dim, atoms, dim, dict);
    //std::cout<<dict->size();

    auto end = std::chrono::steady_clock::now();
    std::cout<<"    # Time Elapsed : "<<std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count()<<" ms"<<std::endl<<std::endl;
}

//******************************************************
//  Update dictionary columns using SVD on Error Matrix
//*****************************************************
void CudaKSvdDenoiser::updateDictionary(){

    int dim = patchSquareDim * patchSquareDim;
    minus<float> binaryOp;
 
    for(int atomIdx = 0 ; atomIdx < sparseCode->m ; atomIdx++){ //->m = # atoms
        std::cout<<"Atom idx:   "<<atomIdx<<std::endl;

        device_vector<int> relevantDataIndices;
        MatrixOps* mult;
        Matrix* dx;
        Matrix* v;
        Matrix* u;
        Matrix* s;
        float bestS;
        buildSvdContainer();

        //Find for each patch relevant atoms --> idx!=0 
		for(int i = 0; i < sparseCode->n; i++){ //-> n = #NoisePatches

			if(sparseCode->deviceVector->data()[(i * sparseCode->m) + atomIdx] != 0) 
				relevantDataIndices.push_back(i); 
		}

        std::cout<<"relevantIndices.size: "<<relevantDataIndices.size()<<std::endl;

        //Only update atom shared by 1 or more pacthes
        if(relevantDataIndices.size()<1)
            continue;

        //Collect input (patches and coeffs) that used this atom
        device_vector<float> selectInput;//relevantDataIndices.size() * dim);
		device_vector<float> selectSparseCode;//relevantDataIndices.size() * sparseCode->m);

        for(int inputIdx : relevantDataIndices) {
			selectInput.insert(selectInput.end(),noisePatches->deviceVector->begin() + inputIdx * dim, noisePatches->deviceVector->begin() + (inputIdx+1) * dim); 
			selectSparseCode.insert(selectSparseCode.end(),sparseCode->deviceVector->begin() + inputIdx * sparseCode->m, sparseCode->deviceVector->begin() + (inputIdx + 1) * sparseCode->m); 
		}

        //Remove atom from dict --> coef at row atomIdx = 0

        for(int i = 0; i < relevantDataIndices.size(); i++)
            selectSparseCode[(i * sparseCode->m) + atomIdx] = 0;

        //DX = Dictionary * selectSparseCode
        mult = MatrixOps::factory(CUBLAS_MULT);
        ((CuBlasMatrixMult*)mult)->setOps(HIPBLAS_OP_N, HIPBLAS_OP_N);
        dx = mult->work(dictionary, new Matrix(sparseCode->m , relevantDataIndices.size(), sparseCode->m, &selectSparseCode));

        std::cout<<"Dopo di moltiplicare"<<std::endl;

        //E = coff - dx
        device_vector<float> error(selectInput.size());

        transform(selectInput.begin(), selectInput.end(), dx->deviceVector->begin(), error.begin(), binaryOp);

        std::cout<<"Prima transform andata"<<std::endl;

        //Compute SVD on E
        svdContainer->setMatrix(new Matrix(dim, relevantDataIndices.size(), dim, &error));
        host_vector<Matrix*> usvt = svdContainer->getDeviceOutputMatrices();

        std::cout<<"SvD ANdato"<<std::endl;
        
        //Traspose V
      /*  tras = MatrixOps::factory(CUBLAS_ADD);
        ((CuBlasMatrixAdd*)tras)->setOps(HIPBLAS_OP_T, HIPBLAS_OP_T);
        v = tras->work(usvt[2], usvt[2]);*/

        //Replace dictionary column
        u = usvt[0];        
        transform(u->deviceVector->begin(), u->deviceVector->begin() + u->m, u->deviceVector->begin(), _1 * -1.f);
        std::cout<<"Seconda transform andata"<<std::endl;
        copy(u->deviceVector->begin(), u->deviceVector->begin() + u->m, dictionary->deviceVector->begin() + atomIdx * dim);

        //Calculate new coeffs
        s = usvt[1];
        v = usvt[2];
        bestS = s->deviceVector->data()[0];
        transform(v->deviceVector->begin(), v->deviceVector->begin() + v->m, v->deviceVector->begin(), _1 * -1.f * bestS);
        std::cout<<"Terza transform andata"<<std::endl;

        for(int i = 0 ; i < relevantDataIndices.size() ; i++ ) {
            int inputIdx = relevantDataIndices[i];
            sparseCode->deviceVector->data()[inputIdx * sparseCode->m + atomIdx] = v->deviceVector->data()[i] ; 
         }

        std::cout<<"FIn iter"<<std::endl;        
    }
}

//******************
//  kSVD algorithm
//  GPU Version
//*************
void CudaKSvdDenoiser::kSvd(){

    for(int i = 0 ; i < iter ; i++){

        std::cout<<"Ksvd-Iter: "<<i+1<<std::endl;

        //OMP phase
        auto start = std::chrono::steady_clock::now();

        CuBlasMatrixOmp* omp = (CuBlasMatrixOmp*) MatrixOps::factory(CUBLAS_OMP);
        sparseCode = omp->work(noisePatches, dictionary);
       // std::cout<<"SparseCode size: "<< sparseCode->deviceVector->size()<<std::endl;
        auto end = std::chrono::steady_clock::now();
        auto tot1 = end - start;
        std::cout<<"    # OMP Time Elapsed : "<<std::chrono::duration_cast<std::chrono::milliseconds>(tot1).count()/1000.<<" s"<<std::endl;

        //Dict update phase
        start = std::chrono::steady_clock::now();
        updateDictionary();
        end = std::chrono::steady_clock::now();
        auto tot2 = end - start;
        std::cout<<"    # Dict update Time Elapsed : "<<std::chrono::duration_cast<std::chrono::milliseconds>(tot2).count()/1000.<<" s"<<std::endl;

        delete sparseCode;
        
        std::cout<<"    # Total time: "<<tot1.count() + tot2.count()<<std::endl<<std::endl; 
    }

    //Compute last Iter sparseCode
    delete sparseCode;

    auto start = std::chrono::steady_clock::now();

    CuBlasMatrixOmp* omp = (CuBlasMatrixOmp*) MatrixOps::factory(CUBLAS_OMP);
    sparseCode = omp->work(noisePatches, dictionary);

    auto end = std::chrono::steady_clock::now();
    auto tot1 = end - start;
    std::cout<<"Last iter OMP Time Elapsed : "<<std::chrono::duration_cast<std::chrono::milliseconds>(tot1).count()<<" ms"<<std::endl;        
}

void CudaKSvdDenoiser::createImage(){

    //Build deNoised Patches
    delete noisePatches;
    
    auto start = std::chrono::steady_clock::now();

    std::cout<<"Matrix mult between Dict and sparseCode";

    CuBlasMatrixMult* mult= (CuBlasMatrixMult*) MatrixOps::factory(CUBLAS_MULT);
    mult->setOps(HIPBLAS_OP_N, HIPBLAS_OP_N);
    noisePatches = mult->work(dictionary, sparseCode);

    auto end = std::chrono::steady_clock::now();
    auto tot1 = end - start;
    std::cout<<"    Time Elapsed : "<<std::chrono::duration_cast<std::chrono::milliseconds>(tot1).count()<<" ms"<<std::endl;


    //Build DenoisedImage
    start = std::chrono::steady_clock::now();

    std::cout<<"Build image denoised";

    device_vector<float>* img = new device_vector<float>(inputMatrix->m * inputMatrix->n);
    device_vector<float> blockWeight(patchSquareDim*patchSquareDim,1);
    device_vector<float> imgWeight(inputMatrix->m * inputMatrix->n);

    int dim = patchSquareDim * patchSquareDim;
    int patchIdx = 0 ;
    for(int i = 0; i + patchSquareDim <= inputMatrix->n; i = i + slidingPatch){
        
        for(int j = 0; j + patchSquareDim <= inputMatrix->m; j = j + slidingPatch){

            int startPatchIdx = i*inputMatrix->m + j ;
            int colIdx = 0;

            device_vector<float> thisPatch(noisePatches->deviceVector->begin() + patchIdx*dim, noisePatches->deviceVector->begin() + (patchIdx + 1)*dim); 

            for(int k = startPatchIdx; k < startPatchIdx + patchSquareDim*inputMatrix->m; k += inputMatrix->m){

                std::transform(thisPatch.begin() + colIdx*patchSquareDim, thisPatch.begin() + (colIdx +1)*patchSquareDim,
                           img->begin() + k, img->begin() + k, std::plus<float>());
                
                std::transform(blockWeight.begin() + colIdx*patchSquareDim ,blockWeight.begin() + (colIdx + 1)*patchSquareDim,
                           imgWeight.begin() + k , imgWeight.begin() + k, std::plus<float>());
            colIdx++ ;
            }
        }
        patchIdx++;
    }

    for(int i = 0 ; i < img->size(); i++)
		img->data()[i] = (inputMatrix->deviceVector->data()[i] + 0.034 * 0.25 * img->data()[i])/(1 + 0.034 * 0.25 * imgWeight[i]); 

    outputMatrix = new Matrix(inputMatrix->m, inputMatrix->n, inputMatrix->m, img);

    end = std::chrono::steady_clock::now();
    tot1 = end - start;

    std::cout<<"    Time Elapsed : "<<std::chrono::duration_cast<std::chrono::milliseconds>(tot1).count()<<" ms"<<std::endl;

}