#include <svdLib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace svd;

void CuSolverGeSvd::init(Matrix* matrix){

    //Call parent method
    SvdCudaEngine::init(matrix);

    //Allocate WorkSpace on device
    hipsolverDnSgesvd_bufferSize(cusolverH, matrix->m, matrix->n, &lWork);
    hipMalloc((void**) &deviceWork , sizeof(double)*lWork);
    
}

void CuSolverGeSvd::work(){

    //DGESVD
    hipsolverDnSgesvd(
        cusolverH,
        'A',
        'A',
        input->m,
        input->n,
        deviceA,
        input->ld,
        deviceS,
        deviceU,
        input->m,
        deviceVT,
        input->n,
        deviceWork,
        lWork,
        deviceRWork,
        deviceInfo
    );
    hipDeviceSynchronize();
}

std::vector<Matrix*> CuSolverGeSvd::getOutputMatrices(){
    hipFree(deviceInfo);
    hipFree(deviceRWork);
    return SvdCudaEngine::getOutputMatrices(); 
}