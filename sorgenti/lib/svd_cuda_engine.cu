#include <svdLib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace svd;

void SvdCudaEngine::init(Matrix* matrix){

    //Call parent method
    SvdEngine::init(matrix);

    //Create cusolverDn handle
    hipsolverDnCreate(&cusolverH) ;

    //Alocate space for cusolverDnInfo
    hipMalloc ((void**)&deviceInfo, sizeof(int));
    
    //Save matrix mem dimension
    size_t space = (matrix->ld)*(matrix->n)*sizeof(float);

    //Allocate memory on device
    //std::cout<<"risultato hipMalloc di A: "<<
    hipMalloc((void**) &deviceA, space);//<<std::endl;
    hipMalloc((void**) &deviceU, (matrix->ld)*(matrix->m)*sizeof(float));
    hipMalloc((void**) &deviceS, (matrix->n)*sizeof(float));
    hipMalloc((void**) &deviceVT, (matrix->n)*(matrix->n)*sizeof(float));

    //Copy matrix on device
    //std::cout<<"risultato memcpy di A: "<<
    hipMemcpy(deviceA, matrix->matrix, space, hipMemcpyHostToDevice);//<<std::endl;

} 

std::vector<Matrix*> SvdCudaEngine::getOutputMatrices(){

    float *hostU, *hostVT, *hostS;
    Matrix *outputU, *outputVT, *outputS;

    //Cpu matrix resource allocation
    hostU = new float[(input->m)*(input->m)]();
    hostVT = new float[(input->n)*(input->n)]();
    hostS = new float[input->n]();

    //Allocate memory on host
    outputU = new Matrix(input->ld, input->m, input->m, hostU);
    outputVT = new Matrix(input->n, input->n, input->n, hostVT);
    outputS = new Matrix (1, input->n, 1, hostS);

    //Copy back to host
    hipMemcpy(hostU, deviceU, (outputU->ld)*(outputU->m)*sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy(hostVT, deviceVT, (outputVT->n)*(outputVT->n)*sizeof(float), hipMemcpyDeviceToHost );
    hipMemcpy(hostS, deviceS, (outputS->n)*sizeof(float), hipMemcpyDeviceToHost);

    //Save SVD
    output = {outputU, outputS, outputVT};

    //Cleaning cuda memory
    hipFree(deviceA);
    hipFree(deviceU);
    hipFree(deviceVT);
    hipFree(deviceS);
    hipFree(deviceWork);
    hipsolverDnDestroy(cusolverH);

    hipDeviceReset();

    return output;
}