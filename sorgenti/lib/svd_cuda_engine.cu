#include <svdLib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace svd;

void SvdCudaEngine::init(Matrix* matrix){

    //Call parent method
    SvdEngine::init(matrix);

    //Create cusolverDn handle
    hipsolverDnCreate(&cusolverH) ;
    
    //Save matrix mem dimension
    size_t space = (matrix->ld)*(matrix->n)*sizeof(double);

    //Allocate memory on device
    hipMalloc((void**) &deviceA, space);
    hipMalloc((void**) &deviceU, (matrix->m)*(matrix->m)*sizeof(double));
    hipMalloc((void**) &deviceS, (matrix->n)*sizeof(double));
    hipMalloc((void**) &deviceVT, (matrix->n)*(matrix->n)*sizeof(double));

    //Copy matrix on device
    hipMemcpy(deviceA, matrix->matrix, space, hipMemcpyHostToDevice);

} 

std::vector<Matrix*> SvdCudaEngine::getOutputMatrices(){

    double *hostU, *hostVT, *hostS;
    Matrix *outputU, *outputVT, *outputS;

    //Cpu matrix resource allocation
    hostU = new double[(input->m)*(input->m)]();
    hostVT = new double[(input->n)*(input->n)]();
    hostS = new double[input->n]();

    //Output matrices
    outputU = new Matrix(input->m, input->m, input->m, hostU);
    outputVT = new Matrix(input->n, input->n, input->n, hostVT);
    outputS = new Matrix (1, input->n, input->n, hostS);

    //Copy back to host
    hipMemcpy(hostU, deviceU, (outputU->ld)*(outputU->n)*sizeof(double), hipMemcpyDeviceToHost );
    hipMemcpy(hostVT, deviceVT, (outputVT->ld)*(outputVT->n)*sizeof(double), hipMemcpyDeviceToHost );
    hipMemcpy(hostS, deviceS, (outputS->ld)*sizeof(double), hipMemcpyDeviceToHost);

    //Save SVD
    output = {outputU, outputS, outputVT};

    //Cleaning cuda memory
    hipFree(deviceA);
    hipFree(deviceU);
    hipFree(deviceVT);
    hipFree(deviceS);
    hipFree(deviceWork);
    hipsolverDnDestroy(cusolverH);

    hipDeviceReset();

    return output;
}