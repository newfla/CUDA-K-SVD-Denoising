#include <svdLib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace svd;

void SvdCudaEngine::init(Matrix* matrix){

    //Call parent method
    SvdEngine::init(matrix);

    //Create cusolverDn handle
    hipsolverDnCreate(&cusolverH) ;

    //Alocate space for cusolverDnInfo
    hipMalloc ((void**)&deviceInfo, sizeof(int));
    
    //Save matrix mem dimension
    size_t space = (matrix->ld)*(matrix->n)*sizeof(double);

    //Allocate memory on device
    //std::cout<<"risultato hipMalloc di A: "<<
    hipMalloc((void**) &deviceA, space);//<<std::endl;
    hipMalloc((void**) &deviceU, (matrix->ld)*(matrix->m)*sizeof(double));
    hipMalloc((void**) &deviceS, (matrix->n)*sizeof(double));
    hipMalloc((void**) &deviceVT, (matrix->n)*(matrix->n)*sizeof(double));

    //Copy matrix on device
    //std::cout<<"risultato memcpy di A: "<<
    hipMemcpy(deviceA, matrix->matrix, space, hipMemcpyHostToDevice);//<<std::endl;

} 

std::vector<Matrix*> SvdCudaEngine::getOutputMatrices(){

    double *hostU, *hostVT, *hostS;
    Matrix *outputU, *outputVT, *outputS;

    //Cpu matrix resource allocation
    hostU = new double[(input->m)*(input->m)]();
    hostVT = new double[(input->n)*(input->n)]();
    hostS = new double[input->n]();

    //Output matrices
    outputU = new Matrix(input->ld, input->m, input->m, hostU);
    outputVT = new Matrix(input->n, input->n, input->n, hostVT);
    outputS = new Matrix (1, input->n, 1, hostS);

    //Copy back to host
    hipMemcpy(hostU, deviceU, (outputU->ld)*(outputU->m)*sizeof(double), hipMemcpyDeviceToHost );
    hipMemcpy(hostVT, deviceVT, (outputVT->n)*(outputVT->n)*sizeof(double), hipMemcpyDeviceToHost );
    hipMemcpy(hostS, deviceS, (outputS->n)*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&infoGpu, deviceInfo, sizeof(int), hipMemcpyDeviceToHost);

    //Save SVD
    output = {outputU, outputS, outputVT};

    //Cleaning cuda memory
    hipFree(deviceA);
    hipFree(deviceU);
    hipFree(deviceVT);
    hipFree(deviceS);
    hipFree(deviceWork);
    hipsolverDnDestroy(cusolverH);

    hipDeviceReset();

    return output;
}