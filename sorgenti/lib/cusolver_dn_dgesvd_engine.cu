#include <svdLib.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>

using namespace svd;

void CuSolverDnDgeSvd::init(Matrix* matrix){

    //Call parent method
    SvdEngine::init(matrix);
    
    //Save matrix mem dimension
    size_t space = (matrix->m)*(matrix->n)*sizeof(double);

    //Allocate memory on device
    hipMalloc((void**) &deviceA, space);

    //Copy matrix on device
    hipMemcpy(deviceA, matrix->matrix, space, hipMemcpyHostToDevice);

}

void CuSolverDnDgeSvd::work(){
    hipFree(deviceA);
}

std::vector<Matrix*> CuSolverDnDgeSvd::getOutputMatrices(){
    return std::vector<Matrix*>();
}