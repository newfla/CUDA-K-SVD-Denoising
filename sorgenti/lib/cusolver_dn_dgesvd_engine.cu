#include <svdLib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace svd;

void CuSolverDnDgeSvd::init(Matrix* matrix){

    //Call parent method
    SvdCudaEngine::init(matrix);

    //Allocate Space on device
    hipsolverDnDgesvd_bufferSize(cusolverH, matrix->m, matrix->n, &lWork);
    hipMalloc ((void**)&deviceInfo, sizeof(int));
    hipMalloc((void**) &deviceWork , sizeof(double)*lWork);


}

void CuSolverDnDgeSvd::work(){

    //DGESVD
    hipsolverDnDgesvd(
        cusolverH,
        'A',
        'A',
        input->m,
        input->n,
        deviceA,
        input->ld,
        deviceS,
        deviceU,
        input->m,
        deviceVT,
        input->n,
        deviceWork,
        lWork,
        deviceRWork,
        deviceInfo
    );
    hipDeviceSynchronize();
    hipMemcpy(&infoGpu, deviceInfo, sizeof(int), hipMemcpyDeviceToHost);

}

std::vector<Matrix*> CuSolverDnDgeSvd::getOutputMatrices(){
    hipFree(deviceInfo);
    hipFree(deviceRWork);
    return SvdCudaEngine::getOutputMatrices(); 
}